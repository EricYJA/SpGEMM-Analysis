#include "hip/hip_runtime.h"
#include <vector>

#include "matrix.cuh"
#include "spmatmul.cuh"

template <typename T_ELEM>
int loadMMSparseMatrix(char *filename, char elem_type, bool csrFormat, int *m,
                       int *n, int *nnz, T_ELEM **aVal, int **aRowInd,
                       int **aColInd, int extendSymMatrix);

__global__ void testMemKernel(CSRMatDevice<float> spmat)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < spmat.m_row_size)
  {
    int s_i = spmat.m_d_rowptr[idx];
    int s_e = spmat.m_d_rowptr[idx + 1];
    for (int i = s_i; i < s_e; ++i)
    {
      printf("(%d, %f) ", spmat.m_d_colidx[i], spmat.m_d_val[i]);
    }
  }
}

void testSetMatData(CSRMatDevice<float> &spmat, std::vector<int> &a_rp_vec, std::vector<int> &a_ci_vec, std::vector<float> &a_va_vec)
{
  for (int i = 0; i < a_rp_vec.size(); ++i)
  {
    spmat.m_d_rowptr[i] = a_rp_vec[i];
  }

  for (int i = 0; i < a_ci_vec.size(); ++i)
  {
    spmat.m_d_colidx[i] = a_ci_vec[i];
    spmat.m_d_val[i] = a_va_vec[i];
  }
}

void testload()
{
  int rowsA = 0; /* number of rows of A */
  int colsA = 0; /* number of columns of A */
  int nnzA = 0;  /* number of nonzeros of A */

  int *h_csrRowPtrA = NULL;
  int *h_csrColIndA = NULL;
  float *h_csrValA = NULL;

  loadMMSparseMatrix<float>("../TestMtx/cage3.mtx", 'd', true, &rowsA,
                            &colsA, &nnzA, &h_csrValA, &h_csrRowPtrA,
                            &h_csrColIndA, true);

  printf("%d, %d, %d\n", rowsA, colsA, nnzA);
}

void testNnz()
{
  std::vector<int> a_rp_vec = {0, 1, 2, 4};
  std::vector<int> a_ci_vec = {1, 2, 0, 1};
  std::vector<float> a_va_vec = {10, 11, 12, 13};

  CSRMatDevice<float> A(3, 3, 4);
  CSRMatDevice<float> B(3, 3, 4);

  testSetMatData(A, a_rp_vec, a_ci_vec, a_va_vec);
  testSetMatData(B, a_rp_vec, a_ci_vec, a_va_vec);

  int nnz_num = countCsrCsrNnzHost<float>(A, B);
  printf("%d\n", nnz_num);

  // int *nnz_num;
  // hipMallocManaged(&nnz_num, sizeof(int));

  // countNnzKernel<float><<<1, 16>>>(A, B, nnz_num);
  // hipDeviceSynchronize();

  // testMemKernel<<<1, 8>>>(A);
  // hipDeviceSynchronize();
}

void testInnPro()
{
  CSRMatDevice<float> A(4, 4, 7);
  CSCMatDevice<float> B(4, 4, 7);
  CSRMatDevice<float> C(4, 4, 49);
  std::vector<u_int> a_rp_vec = {0, 2, 4, 6, 7};
  std::vector<u_int> a_ci_vec = {0, 1, 1, 2, 0, 3, 4};
  std::vector<float> a_va_vec = {1.0, 4.0, 2.0, 3.0, 5.0, 7.0, 8.0};

  std::vector<u_int> b_cp_vec = {0, 2, 4, 6, 7};
  std::vector<u_int> b_ri_vec = {0, 2, 0, 1, 1, 3, 2};
  std::vector<float> b_va_vec = {1.0, 5.0, 4.0, 2.0, 3.0, 9.0, 7.0};

  // hipMemcpy(A.m_d_rowptr, a_rp_vec.data(), (a_rp_vec.size()+1) * sizeof(u_int), hipMemcpyHostToDevice);
  // hipMemcpy(A.m_d_colidx, a_ci_vec.data(), 9 * sizeof(u_int), hipMemcpyHostToHost);
  // hipMemcpy(A.m_d_val, a_va_vec.data(), 9 * sizeof(float), hipMemcpyHostToHost);

  // hipMemcpy(B.m_d_colptr, b_cp_vec.data(), (b_cp_vec.size()+1) * sizeof(u_int), hipMemcpyHostToHost);
  // hipMemcpy(B.m_d_rowidx, b_ri_vec.data(), 9 * sizeof(u_int), hipMemcpyHostToHost);
  // hipMemcpy(B.m_d_val, b_va_vec.data(), 9 * sizeof(float), hipMemcpyHostToHost);

  for (int i = 0; i < 5; ++i)
  {
    A.m_d_rowptr[i] = a_rp_vec[i];
  }

  for (int i = 0; i < 7; ++i)
  {
    A.m_d_colidx[i] = a_ci_vec[i];
    A.m_d_val[i] = a_va_vec[i];
  }

  for (int i = 0; i < 5; ++i)
  {
    B.m_d_colptr[i] = b_cp_vec[i];
  }

  for (int i = 0; i < 7; ++i)
  {
    B.m_d_rowidx[i] = b_ri_vec[i];
    B.m_d_val[i] = b_va_vec[i];
  }

  spgemmInnProMul<float>(A, B, C);

  printf("row ptr:\n");
  for (int i = 0; i < 5; ++i)
  {
    printf("%u, ", C.m_d_rowptr[i]);
  }

  printf("\ncol idx:\n");

  for (int i = 0; i < 9; ++i)
  {
    printf("%u,", C.m_d_colidx[i]);
  }
  printf("\n");
}

int main()
{

  // testNnz();
  testInnPro();
  return 0;
}